#include "hip/hip_runtime.h"
//Image to gray
#include<iostream>

#include<stdio.h>
#include<malloc.h>
#include <hip/hip_runtime.h>
#include <time.h>
//#include <cv.h>
//#include <highgui.h>
#include<opencv2/opencv.hpp>
using namespace std; 
using namespace cv;

#define RED 2
#define GREEN 1
#define BLUE 0

__device__
__host__
unsigned char clamp(int value){
    if(value < 0) value = 0;
    if(value > 255) value = 255;
    return (unsigned char)value;
}

__global__ 
void img2gray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput){

    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if((row < height) && (col < width)){
        imageOutput[row*width+col] = imageInput[(row*width+col)*3+RED]*0.299 + imageInput[(row*width+col)*3+GREEN]*0.587 + imageInput[(row*width+col)*3+BLUE]*0.114;
    }
}   

__global__
void convolutionCU(unsigned char *image, char *mask, int rows, int cols, unsigned char *result){
    
    int i = blockIdx.y*blockDim.y+threadIdx.y;
    int j = blockIdx.x*blockIdx.y+threadIdx.x;
    int sum = 0;

    if(i < rows && j < cols){
        int aux_cols = j - 1, aux_rows = i - 1;
        for(int k = 0; k < 3; k++){ //mask rows
            for(int l = 0; l < 3; l++){
                if(aux_rows >= 0 && aux_cols >= 0 && aux_rows < rows && aux_cols < cols)
                    sum += mask[(k*3)+l]*image[(aux_rows*cols)+aux_cols];
                aux_cols++;
            } 
            aux_rows++;
            aux_cols = j - 1;
        }
        result[(i*cols)+j] = clamp(sum);
    }
}

__global__
void unionCU(unsigned char *imgOutput, unsigned char *Gx, unsigned char *Gy, int rows, int cols){
    int i = blockIdx.y*blockDim.y+threadIdx.y;
    int j = blockIdx.x*blockDim.x+threadIdx.x;

    if(i < rows && j < cols){
        imgOutput[(i * cols) + j] = sqrtf((Gx[(i * cols) + j] * Gx[(i * cols) + j]) + (Gy[(i * cols) + j] * Gy[(i * cols) + j]) );
    }

}

int main(int argc, char **argv){

    if(argc != 2){
  	printf("Is required the name of the image with extension\n");
	return 1;
    }
    
    //For handle errors
    hipError_t error = hipSuccess;
    unsigned char *h_initialImage, *d_initialImage;
    //Image that will be pass to gray
    unsigned char *h_img_gray, *d_img_gray;
    char* imageName = argv[1];
    //Sobel operators
    unsigned char *d_Gx, *d_Gy, *h_G, *d_G, *h_Gx, *h_Gy;
    char *d_XMask, *d_YMask;
    // Image readed
    Mat image;

    image = imread(imageName, 1);

    //Atributes of the image
    Size s = image.size();

    int width = s.width;
    int height = s.height;
    int sz = sizeof(unsigned char)*width*height*image.channels();
    // For the image in gray scale
    int size = sizeof(unsigned char)*width*height;

    // Separte memory for the intial image in host and device
    h_initialImage = (unsigned char*)malloc(sz);
    error = hipMalloc((void**)&d_initialImage,sz);
    if(error != hipSuccess){
        printf("Error asking memory in device for image\n");
        exit(-1);
    }

    // Pass the data to the readed image
    h_initialImage = image.data;

    //Copy data to device
    error = hipMemcpy(d_initialImage,h_initialImage,sz, hipMemcpyHostToDevice);
    if(error != hipSuccess){
        printf("Error copyng the data of h_ImagenInicial to d_ImagenInicial \n");
        exit(-1);
    }

    //Separate memory for gray images in host and device
    h_img_gray = (unsigned char*)malloc(size);
    
    error = hipMalloc((void**)&d_img_gray,size);
    if(error != hipSuccess){
        printf("Error asking memory for d_img_gray\n");
        exit(-1);
    }

    //Block of 32x32 threads = 1024 threads
    dim3 dimBlock(32,32,1);
    dim3 dimGrid(ceil(width/32.0),ceil(height/32.0),1); 
    img2gray<<<dimGrid,dimBlock>>>(d_initialImage, width, height, d_img_gray);
    hipDeviceSynchronize();

   //Copy data of gray image in device to host
/*
    error = hipMemcpy(h_img_gray,d_img_gray,size, hipMemcpyDeviceToHost);
    if(error != hipSuccess){
        printf("Error copyng data of d_img_gray to h_img_gray \n");
        exit(-1);
    }
*/
   //Copy the data of the h_img_gray to the Mat type for save
/*
   Mat res_img_gray;
   res_img_gray.create(height, width, CV_8UC1);
   res_img_gray.data = h_img_gray;
   imwrite("gray_image.png", res_img_gray);
*/
   // --------------- Maks --------------------

   error = hipMalloc((void**)&d_XMask, 3*3*sizeof(int));
   if(error != hipSuccess){
       printf("Error allocating memory for d_XMask\n");
       exit(-1);
   }

   error = hipMalloc((void**)&d_YMask, 3*3*sizeof(int));
   if(error != hipSuccess){
       printf("Error allocating memory for d_YMask\n");
       exit(-1);
   }

   char h_XMask[3*3] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
   char h_YMask[3*3] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};

   error = hipMemcpy(d_XMask, h_XMask, 3*3*sizeof(int), hipMemcpyHostToDevice);
   if(error != hipSuccess){
       printf("Error copying data from h_XMask to d_XMask");
       exit(-1);
   }

   error = hipMemcpy(d_YMask, h_YMask, 3*3*sizeof(int), hipMemcpyHostToDevice);
   if(error != hipSuccess){
       printf("Error copying data from h_YMask to d_YMask");
       exit(-1);
   }

//   ------------------------------ Sobel --------------------------------------------

   h_G = (unsigned char*)malloc(size);

   error = hipMalloc((void**)&d_G, size);

   if(error != hipSuccess){
       printf("Error allocating memory for d_G\n");
       exit(-1);
   }

   h_Gx = (unsigned char*)malloc(size);

   error = hipMalloc((void**)&d_Gx, size);
   if(error != hipSuccess){
       printf("Error allocating memory for d_Gx\n");
       exit(-1);
   }

   h_Gy = (unsigned char*)malloc(size);

   error = hipMalloc((void**)&d_Gy, size);
   if(error != hipSuccess){
       printf("Error allocating memory for d_Gy\n");
       exit(-1);
   }

   //Convolution in Gx
   convolutionCU<<<dimGrid,dimBlock>>>(d_img_gray, d_XMask, height, width, d_Gx);
   hipDeviceSynchronize();

   //Convolution in Gy
   convolutionCU<<<dimGrid,dimBlock>>>(d_img_gray, d_YMask, height, width, d_Gy);
   hipDeviceSynchronize();

   //Union of Gx and Gy results
   unionCU<<<dimGrid,dimBlock>>>(d_G, d_Gx, d_Gy, height, width);
   hipDeviceSynchronize();
   
   error = hipMemcpy(h_G, d_G, size, hipMemcpyDeviceToHost);
   if(error != hipSuccess){
       printf("Error copying data from d_G to h_G\n");
       exit(-1);
   }

 //for test
   error = hipMemcpy(h_Gx, d_Gx, size, hipMemcpyDeviceToHost);
   error = hipMemcpy(h_Gy, d_Gy, size, hipMemcpyDeviceToHost);

   Mat result_Sobel, x_result, y_result;
   result_Sobel.create(height, width, CV_8UC1);
   x_result.create(height, width, CV_8UC1);
   y_result.create(height, width, CV_8UC1);
   result_Sobel.data = h_G;
   x_result.data = h_Gx;
   y_result.data = h_Gy;

   imwrite("sobel_shared.png", result_Sobel);
   imwrite("x_result.png", x_result);
   imwrite("y_result", y_result);
   
   //Free of memory
        
   free(h_img_gray);
   free(h_initialImage);
   free(h_G);
   free(h_Gx);
   free(h_Gy);
   hipFree(d_img_gray);
   hipFree(d_initialImage);    
   hipFree(d_XMask);
   hipFree(d_YMask);
   hipFree(d_Gx);
   hipFree(d_Gy);
   hipFree(d_G);

    return 0;
}
