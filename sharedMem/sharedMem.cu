#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<malloc.h>
#include<opencv2/opencv.hpp>
#include <time.h>
using namespace std;
using namespace cv;

#define RED 2
#define GREEN 1
#define BLUE 0
#define TILE_SIZE 32
#define MAX_MASK_WIDTH 9
#define MASK_WIDTH 3

__constant__ char XM[MASK_WIDTH*MASK_WIDTH];
__constant__ char YM[MASK_WIDTH*MASK_WIDTH];

__device__
__host__
unsigned char clamp(int value){
  if (value < 0) value = 0;
  if (value > 255) value = 255;
  return (unsigned char)value;
}

__host__
void print(unsigned char *M, int rows, int cols){
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < cols; j++) {
      printf("%d ", M[(i * cols) + j]);
    }
    printf("\n");
  }
}

__host__
void convolution(unsigned char *imageInput, int mask[3][3], int rows, int cols, unsigned char *imageOutput){

  for(int i = 0; i < rows; i++) {
    for(int j = 0; j < cols; j++) {
      int sum = 0;
      int aux_cols = j - 1, aux_rows = i - 1;

      for(int k = 0; k < 3; k++) { //mask_rows
        for(int l = 0; l < 3; l++) { //mask_cols
          if ((aux_rows >= 0 && aux_cols >= 0) && (aux_rows < rows && aux_cols < cols))

          sum += mask[k][l]*imageInput[(aux_rows*cols)+ aux_cols];

          aux_cols++;
        }
        aux_rows++;
        aux_cols = j - 1;
      }

      imageOutput[(i * cols) + j] = clamp(sum);
    }
  }
}

__global__
void convolutionCU(unsigned char *imageInput, int *mask, int rows, int cols, unsigned char *imageOutput){

  int i = blockIdx.y*blockDim.y+threadIdx.y;
  int j = blockIdx.x*blockDim.x+threadIdx.x;
  int sum = 0;

  if (i < rows && j < cols) {

    int aux_cols = j - 1, aux_rows = i - 1;
    for (int k = 0; k < 3; k++) {//mask_rows
      for (int l = 0; l < 3; l++) {//mask_cols
        if(aux_rows >= 0 && aux_cols >= 0 && aux_rows < rows && aux_cols < cols)
        sum += mask[(k*3) + l] * imageInput[(aux_rows*cols) + aux_cols];

        aux_cols++;
      }
      aux_rows++;
      aux_cols = j - 1;
    }
    imageOutput[(i * cols) + j] = clamp(sum);
  }
}


__global__
void convolutionShared(unsigned char *imageInput, int height, int width, unsigned char *imageOutput, int Mask_Width, int *Mask){
    int i = blockIdx.y*blockDim.y + threadIdx.y;
    int j = blockIdx.x*blockDim.x + threadIdx.x;

//    __shared__ unsigned char tile[TILE_SIZE + MAX_MASK_WIDTH - 1];
    __shared__ unsigned char tile[32+3-1][32+3-1];
    
    int n = Mask_Width/2;
    int halo_index_up = (blockIdx.x-1)*blockDim.x + threadIdx.x;
    int halo_index_down = (blockIdx.x+1)*blockDim.x + threadIdx.x;
    int halo_index_left = (blockIdx.y-1)*blockDim.y + threadIdx.y;
    int halo_index_right = (blockIdx.y+1)*blockDim.y + threadIdx.y;
    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    int Row = by * TILE_SIZE + ty;
    int Col = bx * TILE_SIZE + tx;

    if(threadIdx.y >= blockDim.y-n){
        tile[threadIdx.y - (blockDim.y-n)][tx] = (halo_index_left < 0) ? 0 : imageInput[halo_index_left];
    }
   if(halo_index_right < n){
        tile[n+blockDim.y+threadIdx.y][tx] = (halo_index_right >= width)? 0 : imageInput[halo_index_right];
    }
    if(threadIdx.x >= blockDim.x-n){
        tile[ty][threadIdx.x-(blockDim.x-n)] = (halo_index_up < 0) ? 0 : imageInput[halo_index_up];
    }
    if(halo_index_down < n){
        tile[ty][n+blockDim.x+threadIdx.x] = (halo_index_down >= height) ? 0 : imageInput[halo_index_down];
    }

   tile[n + ty][n + tx] = imageInput[blockIdx.x*blockDim.x + threadIdx.x];

   __syncthreads();

   float Pvalue = 0;
   for(int m = 0; m < Mask_Width; m++){
       for(int n = 0; n < Mask_Width; n++){
           Pvalue += tile[i][j]*Mask[m*Mask_Width+n];
       }
   }
   
   if(Row < height && Col < width)
	imageOutput[Row*width+Col] = clamp(Pvalue);
   __syncthreads();

}

//--------------------------------------------------------------------------------------------------------------
__global__ void sobelSharedMem(unsigned char *imageInput, int width, int height, unsigned int maskWidth,unsigned char *imageOutput, char *M){
    __shared__ float N_ds[TILE_SIZE + MASK_WIDTH - 1][TILE_SIZE+ MASK_WIDTH - 1];
    int n = maskWidth/2;
    int dest = threadIdx.y*TILE_SIZE+threadIdx.x, destY = dest / (TILE_SIZE+MASK_WIDTH-1), destX = dest % (TILE_SIZE+MASK_WIDTH-1),
        srcY = blockIdx.y * TILE_SIZE + destY - n, srcX = blockIdx.x * TILE_SIZE + destX - n,
        src = (srcY * width + srcX);
    if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
        N_ds[destY][destX] = imageInput[src];
    else
        N_ds[destY][destX] = 0;

    // Second batch loading
    dest = threadIdx.y * TILE_SIZE + threadIdx.x + TILE_SIZE * TILE_SIZE;
    destY = dest /(TILE_SIZE + MASK_WIDTH - 1), destX = dest % (TILE_SIZE + MASK_WIDTH - 1);
    srcY = blockIdx.y * TILE_SIZE + destY - n;
    srcX = blockIdx.x * TILE_SIZE + destX - n;
    src = (srcY * width + srcX);
    if (destY < TILE_SIZE + MASK_WIDTH - 1) {
        if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
            N_ds[destY][destX] = imageInput[src];
        else
            N_ds[destY][destX] = 0;
    }
    __syncthreads();

    int accum = 0;
    int y, x;
    for (y = 0; y < maskWidth; y++)
        for (x = 0; x < maskWidth; x++)
            accum += N_ds[threadIdx.y + y][threadIdx.x + x] * M[y * maskWidth + x];
    y = blockIdx.y * TILE_SIZE + threadIdx.y;
    x = blockIdx.x * TILE_SIZE + threadIdx.x;
    if (y < height && x < width)
        imageOutput[(y * width + x)] = clamp(accum);
    __syncthreads();
}

//--------------------------------------------------------------------------------------------------------------

__global__
void img2grayCU(unsigned char *imageInput, int width, int height, unsigned char *imageOutput){

  int row = blockIdx.y*blockDim.y+threadIdx.y;
  int col = blockIdx.x*blockDim.x+threadIdx.x;

  if((row < height) && (col < width)){

    imageOutput[row*width+col] = imageInput[(row*width+col)*3+RED]*0.299 + imageInput[(row*width+col)*3+GREEN]*0.587
    + imageInput[(row*width+col)*3+BLUE]*0.114;
  }
}


__host__
void Union(unsigned char *imageOutput, unsigned char *Gx, unsigned char *Gy, int rows, int cols){
  for(int i = 0; i < rows; i++){
    for(int j = 0; j < cols; j++){
      imageOutput[(i * cols) + j] = sqrt(pow(Gx[(i * cols) + j],2) + pow(Gx[(i * cols) + j],2));
    }
  }
}


__global__
void UnionCU(unsigned char *imageOutput, unsigned char *Gx, unsigned char *Gy, int rows, int cols){

  int i = blockIdx.y*blockDim.y+threadIdx.y;
  int j = blockIdx.x*blockDim.x+threadIdx.x;

  if (i < rows && j < cols){
    imageOutput[(i * cols) + j] = sqrtf((Gx[(i * cols) + j] * Gx[(i * cols) + j]) + (Gy[(i * cols) + j] * Gy[(i * cols) + j]) );
  }
}


void write(Size s, char* fileName, double elapsedTime){
  long size = s.width * s.height;
  FILE *f = fopen("../global.time", "a");
  if (f == NULL) printf("Error opening file!\n");
  else {
    fprintf(f, "%ld %s %lf\n", size, fileName, elapsedTime);
  }
  fclose(f);
}

int main(int argc, char **argv){

  hipError_t error = hipSuccess;
  clock_t start, end;
  unsigned char *h_imageInput, *d_imageInput, *h_imageGray, *d_imageGray;
  unsigned char *d_Gx, *d_Gy, *h_G, *d_G; // Sobel Operators
  int *d_XMask, *d_YMask;
  char* imageName = argv[1];
  Mat image;

  if (argc != 2) {
    printf("Usage: Image path\n");
    return 1;
  }

  image = imread(imageName, 1);

  if (!image.data) {
    printf("No image Data\n");
    return 1;
  }

  // imshow("Image input", image);
  // waitKey(0);
  //
  // // ------------------------- Gray ------------------------------

  // Timer t("Sobel_Global");
  start = clock();

  Size s = image.size();

  int width = s.width;
  int height = s.height;
  int sz = sizeof(unsigned char) * width * height * image.channels();
  int size = sizeof(unsigned char) * width * height;


  h_imageInput = (unsigned char*)malloc(sz);

  error = hipMalloc((void**)&d_imageInput,sz);
  if (error != hipSuccess) {
    printf("Error allocating memory for d_imageInput\n");
    exit(-1);
  }

  h_imageInput = image.data;

  error = hipMemcpy(d_imageInput, h_imageInput, sz, hipMemcpyHostToDevice);
  if (error != hipSuccess) {
    printf("Error copying data from h_imageInput to d_imageInput\n");
    exit(-1);
  }

  h_imageGray = (unsigned char*)malloc(size);

  error = hipMalloc((void**)&d_imageGray, size);
  if (error != hipSuccess) {
    printf("Error allocating memory for d_imageGray\n");
    exit(-1);
  }

  int blockSize = 32;
  dim3 dimBlock(blockSize, blockSize, 1);
  dim3 dimGrid(ceil(width/float(blockSize)), ceil(height/float(blockSize)), 1);
  img2grayCU<<<dimGrid,dimBlock>>>(d_imageInput, width, height, d_imageGray);
  hipDeviceSynchronize();

  error = hipMemcpy(h_imageGray, d_imageGray, size, hipMemcpyDeviceToHost);
  if (error != hipSuccess) {
    printf("Error copying data from d_imageGray to h_imageGray\n");
    exit(-1);
  }

  //-------------------- Masks -----------------------------
/*
  error = hipMalloc((void**)&d_XMask, 3*3*sizeof(int));
  if (error != hipSuccess) {
    printf("Error allocating memory for d_XMask\n");
    exit(-1);
  }
*/

/*
  error = hipMalloc((void**)&d_YMask, 3*3*sizeof(int));
  if (error != hipSuccess) {
    printf("Error reservando memoria para d_Mascara_Y\n");
    exit(-1);
  }
*/
  int h_XMask[3*3] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
  int h_YMask[3*3] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};

//-------------------- copying to constant memory-------------------------

  error = hipMemcpyToSymbol(HIP_SYMBOL(XM), h_XMask, sizeof(char)*MASK_WIDTH*MASK_WIDTH);
  if(error != hipSuccess){
      printf("Error copying mask h_XMask to M\n");
      exit(-1);
  }

 error = hipMemcpyToSymbol(HIP_SYMBOL(YM), h_YMask, sizeof(char)*MASK_WIDTH*MASK_WIDTH);
  if(error != hipSuccess){
      printf("Error copying mask h_YMask to M\n");
      exit(-1);
  }


//------------------------------------------------------------------------
/*
  error = hipMemcpy(d_XMask, h_XMask, 3*3*sizeof(int), hipMemcpyHostToDevice);
  if (error != hipSuccess) {
    printf("Error copying data from h_XMask to d_XMask\n");
    exit(-1);
  }
*/
/*
  error = hipMemcpy(d_YMask, h_YMask, 3*3*sizeof(int), hipMemcpyHostToDevice);
  if(error != hipSuccess){
    printf("Error copying data from h_YMask to d_YMask\n");
    exit(-1);
  }
*/
  //------------------------ Sobel --------------------------------

  h_G = (unsigned char*)malloc(size);

  error = hipMalloc((void**)&d_G, size);
  if (error != hipSuccess) {
    printf("Error allocating memory for d_G\n");
    exit(-1);
  }

  // Convolution in Gx
  //convolutionCU<<<dimGrid,dimBlock>>>(d_imageGray, d_XMask, height, width, d_Gx);
//  convolutionShared<<<dimGrid,dimBlock>>>(d_imageGray, height, width, d_Gx, 3, d_XMask);
  sobelSharedMem<<<dimGrid,dimBlock>>>(d_imageGray, width, height, MASK_WIDTH, d_Gx, XM);
  hipDeviceSynchronize();

  // Convolution in Gy
//  convolutionCU<<<dimGrid,dimBlock>>>(d_imageGray, d_YMask, height, width, d_Gy);
//  convolutionShared<<<dimGrid,dimBlock>>>(d_imageGray, height, width, d_Gy, 3, d_YMask);
  sobelSharedMem<<<dimGrid,dimBlock>>>(d_imageGray, width, height, MASK_WIDTH, d_Gy, YM);
  hipDeviceSynchronize();

  // Union of Gx and Gy results
  UnionCU<<<dimGrid,dimBlock>>>(d_G, d_Gx, d_Gy, height, width);
  hipDeviceSynchronize();
 

  error = hipMemcpy(h_G, d_G, size, hipMemcpyDeviceToHost);
  if (error != hipSuccess) {
    printf("Error copying data from d_G to h_G\n");
    exit(-1);
  }

  Mat result_Sobel;
  result_Sobel.create(height, width, CV_8UC1);
  result_Sobel.data = h_G;

  // imshow("Sobel CUDA", result_Sobel);
  // waitKey(0);
  imwrite("Sobel_Shared.png", result_Sobel);

  // write(s, imageName, t.elapsed());
  end = clock();
  double time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
  // printf("elapsed time: %lf", time_used);
  write(s, imageName, time_used);

  // free(h_imageInput);
  hipFree(d_imageInput);
  free(h_imageGray);
  hipFree(d_imageGray);
  hipFree(d_XMask);
  hipFree(d_YMask);
  free(h_G);
  hipFree(d_Gx);
  hipFree(d_Gy);
  hipFree(d_G);

  return 0;
}

