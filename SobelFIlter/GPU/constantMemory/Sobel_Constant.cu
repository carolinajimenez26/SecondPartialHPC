#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<malloc.h>
#include<opencv2/opencv.hpp>

using namespace std;
using namespace cv;



#define RED 2
#define GREEN 1
#define BLUE 0

#define MASK_WIDTH 3

//	__constant__ char M[MASK_WIDTH * MASK_WIDTH];

__device__
unsigned char clamp(int value){
    if(value < 0)
        value = 0;
    else
        if(value > 255)
            value = 255;
    return (unsigned char)value;
}

__global__
void convolutionCU(unsigned char *imageInput, int *mask, int rows, int cols, unsigned char *imageOutput){

  int i = blockIdx.y*blockDim.y+threadIdx.y;
  int j = blockIdx.x*blockDim.x+threadIdx.x;
  int sum = 0;

  if (i < rows && j < cols) {

    int aux_cols = j - 1, aux_rows = i - 1;
    for (int k = 0; k < 3; k++) {//mask_rows
      for (int l = 0; l < 3; l++) {//mask_cols
        if(aux_rows >= 0 && aux_cols >= 0 && aux_rows < rows && aux_cols < cols)
        sum += mask[(k*3) + l] * imageInput[(aux_rows*cols) + aux_cols];

        aux_cols++;
      }
      aux_rows++;
      aux_cols = j - 1;
    }
    imageOutput[(i * cols) + j] = clamp(sum);
  }
}

__global__
void img2grayCU(unsigned char *imageInput, int width, int height, unsigned char *imageOutput){

  int row = blockIdx.y*blockDim.y+threadIdx.y;
  int col = blockIdx.x*blockDim.x+threadIdx.x;

  if((row < height) && (col < width)){

    imageOutput[row*width+col] = imageInput[(row*width+col)*3+RED]*0.299 + imageInput[(row*width+col)*3+GREEN]*0.587
    + imageInput[(row*width+col)*3+BLUE]*0.114;
  }
}


__global__
void UnionCU(unsigned char *imageOutput, unsigned char *Gx, unsigned char *Gy, int rows, int cols){

  int i = blockIdx.y*blockDim.y+threadIdx.y;
  int j = blockIdx.x*blockDim.x+threadIdx.x;

  if (i < rows && j < cols){
    imageOutput[(i * cols) + j] = sqrtf((Gx[(i * cols) + j] * Gx[(i * cols) + j]) + (Gx[(i * cols) + j] * Gx[(i * cols) + j]) );
  }
}


int main(int argc, char const *argv[])
{


  if (argc != 2) {
  	printf("Usage: Image path\n");
  	return 1;
	}

	///////////////////////declaracion de variables ////////////////////////////


	hipError_t error = hipSuccess;

	//times
	clock_t start, end;
  	double time_used;
  	char* imageName = argv[1];

  	//imagen inicial
  	unsigned char *h_ImageInit;
  	unsigned char *d_ImageInit;

  	//imagen en grises
  	unsigned char *d_imageGray;
  	//unsigned char *h_imageGray;

  	//imagenes con filtro en X y en Y
  	unsigned char *d_Gx, *d_Gy;

  	//imagen final
  	unsigned char *h_G, *d_G; 

  	//mascaras device
  	int *d_XMask, *d_YMask;

  	//mascaras device
  	int h_XMask[MASK_WIDTH*MASK_WIDTH] = {-1, 0, 1,-2, 0, 2,-1, 0, 1};
  	int h_YMask[MASK_WIDTH*MASK_WIDTH] = {-1,-2,-1, 0, 0, 0, 1, 2, 1};

  	//int h_XMask[3*3] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
  	//int h_YMask[3*3] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};

  	char* imageName = argv[1];


  	//carga la imagen inicial
  	Mat image;
  	image = imread(imageName, 1);

	if (!image.data) {
		printf("No image Data\n");
	    return 1;
	}

	//se toman los parametros de la imagen
	Size s = image.size();
	int width = s.width;
	int height = s.height;

	int size = sizeof(unsigned char) * width * height * image.channels();
  	int sizeGray = sizeof(unsigned char) * width * height;


  	///////////reserve memory for Host and device ///////////////////////////


	//Imagen inicial en el Host
	h_ImageInit = (unsigned char*)malloc(size);
	//imagen final  host
	h_G = (unsigned char*)malloc(size);



	///////////////////// hipMalloc ////////////////////////////////////////
	//imagen inicial device
	error = hipMalloc((void**)&d_ImageInit,size);
  	if (error != hipSuccess) {
    	printf("Error allocating memory for d_imageInput\n");
    	exit(-1);
  	}

  	//imagen en grises device
  	 error = hipMalloc((void**)&d_imageGray, size);
  if (error != hipSuccess) {
    printf("Error allocating memory for d_imageGray\n");
    exit(-1);
  }

  //Mascara en x 
  error = hipMalloc((void**)&d_XMask, 3*3*sizeof(int));
  if (error != hipSuccess) {
    printf("Error allocating memory for d_XMask\n");
    exit(-1);
  }

  //mascara en Y
  error = hipMalloc((void**)&d_YMask, 3*3*sizeof(int));
  if (error != hipSuccess) {
    printf("Error reservando memoria para d_Mascara_Y\n");
    exit(-1);
  }

  //convoluciones//

  //imagen convolucion Gx device
  error = hipMalloc((void**)&d_Gx, size);
  if (error != hipSuccess) {
    printf("Error allocating memory for d_Gx\n");
    exit(-1);
  }

  //imagen convolucion Gy device
  error = hipMalloc((void**)&d_Gy, size);
  if (error != hipSuccess) {
    printf("Error allocating memory for d_Gy\n");
    exit(-1);
  }


  //imagen final en device Union
  error = hipMalloc((void**)&d_G, size);
  if (error != hipSuccess) {
    printf("Error allocating memory for d_G\n");
    exit(-1);
  }

  /////////////////CudaMemCpy//////////////////////////////////////////////


  //carga la imagen inicial
  h_imageInput = image.data;

  error = hipMemcpy(d_imageInput, h_imageInput, sz, hipMemcpyHostToDevice);
  if (error != hipSuccess) {
    printf("Error copiando  imagen inicial de host a device\n");
    exit(-1);
  }


 //mascaras
  error = hipMemcpy(d_XMask, h_XMask, 3*3*sizeof(int), hipMemcpyHostToDevice);
  if (error != hipSuccess) {
    printf("Error copiando mascara X  de host a device\n");
    exit(-1);
  }

  error = hipMemcpy(d_YMask, h_YMask, 3*3*sizeof(int), hipMemcpyHostToDevice);
  if(error != hipSuccess){
    printf("Error copiando mascara Y  de host a device\n");
    exit(-1);
  }

  //////////////////////////////Grises//////////////////////////////////////

  int blockSize = 32;
  dim3 dimBlock(blockSize, blockSize, 1);
  dim3 dimGrid(ceil(width/float(blockSize)), ceil(height/float(blockSize)), 1);
  img2grayCU<<<dimGrid,dimBlock>>>(d_ImageInit, width, height, d_imageGray);
  hipDeviceSynchronize();

  ////////////////////////////Convoluciones//////////////////////////////////

  // Convolution in Gx
  convolutionCU<<<dimGrid,dimBlock>>>(d_imageGray, d_XMask, height, width, d_Gx);
  hipDeviceSynchronize();

  // Convolution in Gy
  convolutionCU<<<dimGrid,dimBlock>>>(d_imageGray, d_YMask, height, width, d_Gy);
  hipDeviceSynchronize();


  // Union of Gx and Gy  ///// Sobel
  UnionCU<<<dimGrid,dimBlock>>>(d_G, d_Gx, d_Gy, height, width);
  hipDeviceSynchronize();

  //Resultado de
  error = hipMemcpy(h_G, d_G, size, hipMemcpyDeviceToHost);
  if (error != hipSuccess) {
    printf("Error copiando resultado  del device al host\n");
    exit(-1);
  }


  //crea la imagen resultante
  Mat result_Sobel;
  result_Sobel.create(height, width, CV_8UC1);
  result_Sobel.data = h_G;

  imwrite("Sobel_const.jpg", result_Sobel);

  //liberar memoria

  free(h_imageInput);
  //free(h_imageGray);
  free(h_G);


  hipFree(d_imageInput);  
  hipFree(d_imageGray);
  hipFree(d_XMask);
  hipFree(d_YMask);  
  hipFree(d_Gx);
  hipFree(d_Gy);
  hipFree(d_G);


	return 0;
}